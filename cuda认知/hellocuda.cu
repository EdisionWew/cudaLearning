
#include <hip/hip_runtime.h>
#include <stdio.h>

//编辑kernel function

__global__ void helloFromGpu(void)
{
    printf("hello cudaGpu!\n");
}

int main()
{
	printf("hello world from CPU!\n");
	helloFromGpu<<<1,10>>>();
	hipDeviceReset();//该段代码重置化gpu，用来显示催毁cuda程序占用的gpu内存资源
	return 0;
}
